#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <cmath>

#include "BLAS/axpy.h"
#include "/usr/local/cuda-12.3/targets/x86_64-linux/include/hip/hip_runtime.h"

int main()
{
	// Problem variables
	std::vector<float> a;
	std::vector<float> x;
	float y = 1.0;
	size_t N = 1000;

	// device side
	float* dev_a;
	float* dev_x;

	// Problem setup
	a.resize(N);
	x.resize(N);
    for (int i=0; i<N; i++)
    {
        a[i] = 1;
        x[i] = (float) i;
    }

    hipSetDevice(0);
	// copy to device
	hipMalloc((void**) &dev_a, N * sizeof(float));
	hipMalloc((void**) &dev_x, N * sizeof(float));

	hipMemcpy(a.data(), dev_a, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(x.data(), dev_x, N * sizeof(float), hipMemcpyHostToDevice);


	// call kernel
	size_t n_threads = 512;
	size_t n_blocks = std::ceil(N / n_threads);
	axpy<<<n_blocks, n_threads>>>(dev_a, dev_x, y, N);

	//copy back
	hipMemcpy(dev_x, x.data(), N * sizeof(float), hipMemcpyDeviceToHost);

    for (auto& value : x)
    {
        std::cout << value << " ";
    }

	return 0;
}